#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Device/Util/Timer.cuh>
#include "Static/CommonNeighbors/commonNeigh.cuh"
#include <iostream>
#include <fstream>
#include <math.h> 

namespace hornets_nest {

commonNeigh::commonNeigh(HornetGraph& hornet) :
                                       StaticAlgorithm(hornet)

{                                       
}

commonNeigh::~commonNeigh(){
    release();
}
/*
struct OPERATOR_InitPairCommonNeighCounts {
    triangle_t *d_countsPerPair;
    // usage in ForAllEdges preferable, but unclear how to get dst index in adjacency
    OPERATOR (Vertex &vertex) {
        degree_t degree = vertex.degree();
        for (int i=0; i<degree; i++) {
            eoff_t src_offset = d_offsets[vertex.id()];
            //d_countsPerPair[src_offset+i] = 1; // test: total count should = |E|
            d_countsPerPair[src_offset+i] = 0; 
        }
    }
};
*/

/*
 * Search for position of key in array
 */
__device__ __forceinline__
void indexBinarySearch(vid_t* data, vid_t arrLen, vid_t key, vid_t& pos) {
    vid_t low = 0;
    vid_t high = arrLen - 1;
    while (high >= low) {
        vid_t middle = (low + high) / 2;
        if (data[middle] == key) {
             pos = middle;
             return;
        } else if (data[middle] < key) {
            low = middle + 1;
		} else {
            high = middle - 1;
		}
    }
}

struct OPERATOR_AdjIntersectionCountBalanced {
    triangle_t* d_countsPerPair;
    const unsigned int vertex_offset;
    const unsigned int nV;

    OPERATOR(Vertex &u, Vertex& v, vid_t* ui_begin, vid_t* ui_end, vid_t* vi_begin, vid_t* vi_end, int FLAG) {
        int count = 0;
        if (!FLAG) {
            int comp_equals, comp1, comp2, ui_bound, vi_bound;
            //printf("Intersecting %d, %d: %d -> %d, %d -> %d\n", u.id(), v.id(), *ui_begin, *ui_end, *vi_begin, *vi_end);
            while (vi_begin <= vi_end && ui_begin <= ui_end) {
                comp_equals = (*ui_begin == *vi_begin);
                count += comp_equals;
                comp1 = (*ui_begin >= *vi_begin);
                comp2 = (*ui_begin <= *vi_begin);
                ui_bound = (ui_begin == ui_end);
                vi_bound = (vi_begin == vi_end);
                // early termination
                if ((ui_bound && comp2) || (vi_bound && comp1))
                    break;
                if ((comp1 && !vi_bound) || ui_bound)
                    vi_begin += 1;
                if ((comp2 && !ui_bound) || vi_bound)
                    ui_begin += 1;
            }
        } else {
            vid_t vi_low, vi_high, vi_mid;
            while (ui_begin <= ui_end) {
                auto search_val = *ui_begin;
                vi_low = 0;
                vi_high = vi_end-vi_begin;
                while (vi_low <= vi_high) {
                    vi_mid = (vi_low+vi_high)/2;
                    auto comp = (*(vi_begin+vi_mid) - search_val);
                    if (!comp) {
                        count += 1;
                        break;
                    }
                    if (comp > 0) {
                        vi_high = vi_mid-1;
                    } else if (comp < 0) {
                        vi_low = vi_mid+1;
                    }
                }
                ui_begin += 1;
            }
        }
        //printf("(%d, %d)\n", u.id(), v.id());
        // NOTE: this will error if u > v
        bool sourceSmaller = u.id() < v.id();
        vid_t u_id = sourceSmaller ? u.id() : v.id();
        vid_t v_id = sourceSmaller ? v.id() : u.id(); 
        eoff_t offset = (u_id-vertex_offset)*nV+v_id;
        atomicAdd(d_countsPerPair+offset, count);
    }
};


triangle_t commonNeigh::countTriangles(){

    triangle_t* h_countsPerPair;
    host::allocate(h_countsPerPair, hornet.nE());
    gpu::copyToHost(d_countsPerPair, hornet.nE(), h_countsPerPair);
    triangle_t sum=0;
    for(int i=0; i<hornet.nE(); i++){
        // printf("%d %ld\n", i,outputArray[i]);
        sum+=h_countsPerPair[i];
    }
    free(h_countsPerPair);
    //triangle_t sum=gpu::reduce(hd_triangleData().countsPerPair, hd_triangleData().nv+1);

    return sum;
}

/*
 * Writes common neighbor counts to file
 */
void commonNeigh::writeToFile(char* outPath) {

    triangle_t* h_countsPerPair;
    host::allocate(h_countsPerPair, hornet.nE());
    gpu::copyToHost(d_countsPerPair, hornet.nE(), h_countsPerPair);

	std::ofstream fout;
    fout.open(outPath);
	const eoff_t* offsets = hornet.csr_offsets();
	const vid_t* edges = hornet.csr_edges();
	vid_t dst = -1;
    fout << "# Nodes: " << hornet.nV() << " Edges: " << hornet.nE() << std::endl;
	triangle_t triangles = 0;
    for (vid_t src=0; src<hornet.nV(); src++) {
		for (eoff_t j=offsets[src]; j<offsets[src+1]; j++) {
			dst = edges[j];
			triangles = h_countsPerPair[j];
			fout << src << " " << dst << " " << triangles << std::endl;
		}
    }
    fout.close();
    free(h_countsPerPair);
}

void printResults(triangle_t* d_countsPerPair, unsigned int vStart, unsigned int vEnd, unsigned int nV) {
    
    triangle_t* h_countsPerPair;
    host::allocate(h_countsPerPair, (vEnd-vStart)*nV);
    gpu::copyToHost(d_countsPerPair, (vEnd-vStart)*nV, h_countsPerPair);
    for (unsigned int v = vStart; v < vEnd; v++) {
        for (int i=0; i<nV; i++) {
            std::cout << "(" << v << "," << i << "): " << h_countsPerPair[(v-vStart)*nV+i] << std::endl; 
        }
    }
}

void commonNeigh::reset(){
    //forAllVertices(hornet, OPERATOR_InitTriangleCounts { countsPerPair, hornet.device_csr_offsets() });
}

void commonNeigh::run() {
    return;
}

void commonNeigh::run(const int WORK_FACTOR=1){
  
    using namespace timer;
    const unsigned int nV = hornet.nV(); 
    const unsigned int QUEUE_PAIRS_LIMIT = min(nV*nV, (int)5E8); // allocate memory for pairs up to limit
    std::cout << "QUEUE_PAIRS_LIMIT: " << QUEUE_PAIRS_LIMIT << std::endl;
    vid2_t* vertexPairs = NULL;
    const unsigned int vStepSize = floor((double)QUEUE_PAIRS_LIMIT/nV); // double to avoid underflow from division
    std::cout << "vStepSize: " << vStepSize << std::endl;
    unsigned int vStart = 0;
    unsigned int vEnd = min(vStart + vStepSize, nV); 
    unsigned int queue_size;

    vertexPairs = new vid2_t[QUEUE_PAIRS_LIMIT];
    vid2_t* d_vertexPairs = nullptr; 
    gpu::allocate(d_vertexPairs, QUEUE_PAIRS_LIMIT); // could be smaller
    gpu::allocate(d_countsPerPair, vStepSize*nV);
    hipMemset(d_countsPerPair, 0, vStepSize*nV*sizeof(triangle_t)); // initialize pair common neighbor counts to 0
    Timer<DEVICE> TM(5);
    while (vStart < nV) {
        std::cout << "vStart: " << vStart << ", " << "vEnd: " << vEnd << std::endl;
       // fill array 
       TM.start();
       for (unsigned int v = vStart; v < vEnd; v++) {
           for (unsigned int index = 0; index < nV; index++) {
              vertexPairs[(v-vStart)*nV+index] = xlib::make2<vid_t>(v, index);
           }
       }
       queue_size = (vEnd-vStart)*nV;
       std::cout << "queue_size: " << queue_size << std::endl;
       hipMemcpy(d_vertexPairs, vertexPairs, queue_size*sizeof(vid2_t), hipMemcpyHostToDevice);
       TM.stop();
       TM.print("Creating pairs:");
       forAllAdjUnions(hornet, d_vertexPairs, queue_size, OPERATOR_AdjIntersectionCountBalanced { d_countsPerPair, vStart, nV }, WORK_FACTOR); 
       printResults(d_countsPerPair, vStart, vEnd, nV);
       vStart = vEnd;
       vEnd = min(vEnd+vStepSize, nV);
       hipMemset(d_countsPerPair, 0, vStepSize*nV*sizeof(triangle_t)); // initialize pair common neighbor counts to 0
    }

    delete [] vertexPairs;
    gpu::free(d_vertexPairs);
}


void commonNeigh::release(){
    gpu::free(d_countsPerPair);
    d_countsPerPair = nullptr;
}

void commonNeigh::init(){
    //gpu::allocate(countsPerPair, hornet.nE());
    reset();
}

} // namespace hornets_nest
